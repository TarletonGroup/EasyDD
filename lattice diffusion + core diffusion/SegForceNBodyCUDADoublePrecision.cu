/**************************************************************************
 *
 *      Module:  This module contains the functions needed for
 *               calculating interactions between dislocation
 *               segments.  See Tom Arsenlis for details on the
 *               method used to do the calculations.
 *
 *      Included functions:
 *               SegSegForceIntegrals()
 *               SpecialSegSegForce()
 *               SegSegForce()
 *               LocalSegForces()
 *               ComputeForces()
 *
 *************************************************************************/
//#include <mex.h>
//#include <cuda.h>
//#include <cuda_runtime.h>


#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>
#include <stdio.h>

/**************************************************************************
 * The following section defines the variables used later on:
 *
 *          p1*,p2*      endpoints for dislocation segment beginning
 *                       at point p1 and ending at point p2
 *          p3*,p4*      endpoints for dislocation segment beginning
 *                       at point p3 and ending at point p4
 *          bpx,bpy,bpz  burgers vector for segment p1->p2
 *          bx,by,bz     burgers vector for segment p3->p4
 *          a            core parameter
 *          MU           shear modulus
 *          NU           poisson ratio
 *          seg12Local   1 if either node of segment p1->p2 is local to
 *                       the current domain, zero otherwise.
 *          seg34Local   1 if either node of segment p3->p4 is local to
 *                       the current domain, zero otherwise.
 *          fp1*, fp2*,  pointers to locations in which to return forces
 *          fp3*, fp4*   on nodes p1 thru p4 respectively
 *
 *
 *          With regards to other variables, the following notation has
 *          used: t stands for the line vector, p stands for the normalised
 *          of a vector, c stands for a cross product operation, d stands 
 *          for a dot product operation. For example the variable,
 *          
 *          tctpctx is "t" "c"ross product with "t" "p"rime "c"ross product
 *                      with "t" "x" component
 *
 *************************************************************************/

__global__ void SegForceNBodyCUDA(double const *SoA,
                    double const a, double const MU, double const NU,
                    int const S,
                    double *f0x, double *f0y, double *f0z,
                    double *f1x, double *f1y, double *f1z);

__device__ void SegSegForce(double p1x, double p1y, double p1z,
                        double p2x, double p2y, double p2z,
                        double p3x, double p3y, double p3z,
                        double p4x, double p4y, double p4z,
                        double bpx, double bpy, double bpz,
                        double bx, double by, double bz,
                        double a, double MU, double NU,
                        int seg12Local, int seg34Local,
                        double *fp1x, double *fp1y, double *fp1z,
                        double *fp2x, double *fp2y, double *fp2z,
                        double *fp3x, double *fp3y, double *fp3z,
                        double *fp4x, double *fp4y, double *fp4z);

__device__ void SpecialSegSegForce(double p1x, double p1y, double p1z,
                               double p2x, double p2y, double p2z,
                               double p3x, double p3y, double p3z,
                               double p4x, double p4y, double p4z,
                               double bpx, double bpy, double bpz,
                               double bx, double by, double bz,
                               double a, double MU, double NU, double ecrit,
                               int seg12Local, int seg34Local,
                               double *fp1x, double *fp1y, double *fp1z,
                               double *fp2x, double *fp2y, double *fp2z,
                               double *fp3x, double *fp3y, double *fp3z,
                               double *fp4x, double *fp4y, double *fp4z);

__global__ void SegForceNBodyCUDA(double const *SoA,
                    double const a, double const MU, double const NU,
                    int const S,
                    double *f0x, double *f0y, double *f0z,
                    double *f1x, double *f1y, double *f1z)
{
    int j;
    double p1x, p1y, p1z;
    double p2x, p2y, p2z;
    double p3x, p3y, p3z;
    double p4x, p4y, p4z;
    double b1x, b1y, b1z;
    double b2x, b2y, b2z;
    int seg12Local, seg34Local;
    double fp1x, fp1y, fp1z;
    double fp2x, fp2y, fp2z;
    double fp3x, fp3y, fp3z;
    double fp4x, fp4y, fp4z;
        
        seg12Local=1;

        //pure N-body formulation. only calculated fp1 and fp2, and call function N^2 times.
        
        int i = threadIdx.x + blockIdx.x * blockDim.x;

        if (i < S) {
            
            b1x = SoA[9*i];
            b1y = SoA[9*i+1];
            b1z = SoA[9*i+2];
            p1x = SoA[9*i+3];
            p1y = SoA[9*i+4];
            p1z = SoA[9*i+5];
            p2x = SoA[9*i+6];
            p2y = SoA[9*i+7];
            p2z = SoA[9*i+8];

            for (j=0; j<S; j++) { //use blocks of B_vec, P1_vec, P2_vec in shared memory for efficient data reuse?

                    b2x = SoA[9*j];
                    b2y = SoA[9*j+1];
                    b2z = SoA[9*j+2];
                    p3x = SoA[9*j+3];
                    p3y = SoA[9*j+4];
                    p3z = SoA[9*j+5];
                    p4x = SoA[9*j+6];
                    p4y = SoA[9*j+7];
                    p4z = SoA[9*j+8];
                    
                    SegSegForce(p1x, p1y, p1z,
                                p2x, p2y, p2z,
                                p3x, p3y, p3z,
                                p4x, p4y, p4z,
                                b1x, b1y, b1z,
                                b2x, b2y, b2z,
                                a, MU, NU,
                                seg12Local, seg34Local,
                                &fp1x, &fp1y, &fp1z,
                                &fp2x, &fp2y, &fp2z,
                                &fp3x, &fp3y, &fp3z,
                                &fp4x, &fp4y, &fp4z);
                    
                    f0x[i] += fp1x;
                    f0y[i] += fp1y;
                    f0z[i] += fp1z;
                    
                    f1x[i] += fp2x;
                    f1y[i] += fp2y;
                    f1z[i] += fp2z;
            }
        }
    return;
}

/*
 *      SpecialSegSegForce() and SegSegForce() each reference the
 *      other function, so we need prototypes for these functions...
 */

/*-------------------------------------------------------------------------
 *
 *      Function:     SpecialSegSegForceIntegrals
 *      Description:  Calculates the integrals required for the
 *                    force calculation for near-parallel dislocation
 *                    segment pairs.
 *-----------------------------------------------------------------------*/
__device__ void SpecialSegSegForceIntegrals(double a2,double d2,double yin,double zin,
                                  double *f_003, double *f_103, double *f_013,
                                 double *f_113, double *f_213, double *f_123,
                                 double *f_005, double *f_105, double *f_015,
                                 double *f_115, double *f_215, double *f_125)
{
        double a2_d2, a2d2inv, ypz, ymz, Ra, Rainv, Log_Ra_ypz, common1;

        a2_d2 = a2 + d2;   
        a2d2inv = 1 / a2_d2;
        ypz = yin + zin;
        ymz = yin - zin;
        Ra = sqrt(a2_d2 + ypz*ypz);
        Rainv = 1 / Ra;
        Log_Ra_ypz = log(Ra + ypz);
        
        common1 = ymz * Ra * a2d2inv;

        *f_003 = Ra * a2d2inv;
        *f_103 = -0.5 * (Log_Ra_ypz - common1);
        *f_013 = -0.5 * (Log_Ra_ypz + common1);
        *f_113 = -Log_Ra_ypz;
        *f_213 = zin*Log_Ra_ypz - Ra;
        *f_123 = yin*Log_Ra_ypz - Ra;
        
        *f_005 =  a2d2inv * (2*a2d2inv*Ra - Rainv);
        *f_105 =  a2d2inv * (common1 - yin*Rainv);
        *f_015 = -a2d2inv * (common1 + zin*Rainv);
        *f_115 = -a2d2inv * ypz * Rainv;
        *f_215 =  Rainv - zin * *f_115;
        *f_125 =  Rainv - yin * *f_115;

        return;
}


/*-------------------------------------------------------------------------
 *
 *      Function:     SegSegForceIntegrals
 *      Description:  Calculates the integrals required for the
 *                    force calculation
 *
 *-----------------------------------------------------------------------*/
__device__ void SegSegForceIntegrals(double a, double d, double c,double yin,double zin,
                        double *f_003, double *f_103, double *f_013,
                        double *f_113, double *f_203, double *f_023,
                        double *f_005, double *f_105, double *f_015,
                        double *f_115, double *f_205, double *f_025,
                        double *f_215, double *f_125, double *f_225,
                        double *f_305, double *f_035, double *f_315,
                        double *f_135)
{
        double c2, onemc2, onemc2inv, denom;
        double a2_d2, y2, z2, Ra;
        double Ra_Rdot_t, log_Ra_Rdot_t, zlog_Ra_Rdot_t;
        double Ra_Rdot_tp, log_Ra_Rdot_tp, ylog_Ra_Rdot_tp;
        double Rainv, Ra2_R_tinv, zRa2_R_tinv, z2Ra2_R_tinv;
        double Ra2_R_tpinv, yRa2_R_tpinv, y2Ra2_R_tpinv;
        double adf_003, tf_113;
        double commonf025, commonf035, commonf205;
        double commonf223, commonf225, commonf305;
        double ycommonf025, zcommonf305, zcommonf205;

        c2 = c*c;
        onemc2 = 1-c2; 
        onemc2inv = 1/onemc2;         
        a2_d2 = a*a+d*d*onemc2;
        y2    = yin*yin;
        z2    = zin*zin;
        Ra    = sqrt(a2_d2 + y2 + z2 + 2*yin*zin*c);       
        Rainv = 1/Ra;
        
        Ra_Rdot_tp = Ra+zin+yin*c;       
        Ra_Rdot_t  = Ra+yin+zin*c;       
         
         log_Ra_Rdot_tp =     log(Ra_Rdot_tp); 
        ylog_Ra_Rdot_tp = yin*log_Ra_Rdot_tp;
        
         log_Ra_Rdot_t =     log(Ra_Rdot_t); 
        zlog_Ra_Rdot_t = zin*log_Ra_Rdot_t; 
        
          Ra2_R_tpinv = Rainv/Ra_Rdot_tp; 
         yRa2_R_tpinv = yin*  Ra2_R_tpinv;
        y2Ra2_R_tpinv = yin* yRa2_R_tpinv;
        
          Ra2_R_tinv = Rainv/Ra_Rdot_t; 
         zRa2_R_tinv = zin* Ra2_R_tinv;
        z2Ra2_R_tinv = zin*zRa2_R_tinv;
        
        denom = 1/sqrt(onemc2*a2_d2);
        *f_003 = -2*denom*atan((1+c)*(Ra+yin+zin)*denom);
        
        adf_003 = a2_d2**f_003;
        commonf223 = (c*Ra - adf_003)*onemc2inv;

        *f_103 = (c*log_Ra_Rdot_t  - log_Ra_Rdot_tp)*onemc2inv; 
        *f_013 = (c*log_Ra_Rdot_tp - log_Ra_Rdot_t )*onemc2inv;
        *f_113 = (c*adf_003 - Ra)*onemc2inv;
        *f_203 =  zlog_Ra_Rdot_t  + commonf223;
        *f_023 =  ylog_Ra_Rdot_tp + commonf223;

         commonf225 = *f_003 - c*Rainv;
         commonf025 = c*yRa2_R_tpinv - Rainv;
        ycommonf025 = yin*commonf025;
         commonf205 = c*zRa2_R_tinv  - Rainv;
        zcommonf205 = zin*commonf205;
         commonf305 = log_Ra_Rdot_t  -(yin-c*zin)*Rainv - c2*z2Ra2_R_tinv;
        zcommonf305 = zin*commonf305;
         commonf035 = log_Ra_Rdot_tp -(zin-c*yin)*Rainv - c2*y2Ra2_R_tpinv;

        tf_113 = 2**f_113;
        
        *f_005 = (*f_003 - yRa2_R_tpinv - zRa2_R_tinv)/(a2_d2);
        *f_105 = (Ra2_R_tpinv - c*Ra2_R_tinv)*onemc2inv;
        *f_015 = (Ra2_R_tinv  - c*Ra2_R_tpinv)*onemc2inv;
        *f_115 = (Rainv - c*(yRa2_R_tpinv + zRa2_R_tinv + *f_003))*onemc2inv;
        *f_205 = (yRa2_R_tpinv + c2*zRa2_R_tinv  + commonf225)*onemc2inv;
        *f_025 = (zRa2_R_tinv  + c2*yRa2_R_tpinv + commonf225)*onemc2inv;
        *f_215 = (*f_013 - ycommonf025 + c*(zcommonf205-*f_103))*onemc2inv; 
        *f_125 = (*f_103 - zcommonf205 + c*(ycommonf025 - *f_013))*onemc2inv; 
        *f_225 = (*f_203 - zcommonf305 + c*(y2*commonf025 - tf_113))*onemc2inv;
        *f_305 = (y2Ra2_R_tpinv + c*commonf305 + 2**f_103)*onemc2inv;
        *f_035 = (z2Ra2_R_tinv  + c*commonf035 + 2**f_013)*onemc2inv;
        *f_315 = (tf_113 - y2*commonf025 + c*(zcommonf305 - *f_203))*onemc2inv;
        *f_135 = (tf_113 - z2*commonf205 + c*(yin*commonf035-*f_023))*onemc2inv;
       
        return;
}


/*-------------------------------------------------------------------------
 *
 *      Function:     SpecialSegSegForce
 *      Description:  Special function for calculating forces between
 *                    dislocation segments too close to parallel to be
 *                    calculated via the function used for regular
 *                    segment/segment forces.
 *      Arguments:
 *          p1*,p2*      endpoints for dislocation segment beginning
 *                       at point p1 and ending at point p2
 *          p3*,p4*      endpoints for dislocation segment beginning
 *                       at point p3 and ending at point p4
 *          bpx,bpy,bpz  burgers vector for segment p1->p2
 *          bx,by,bz     burgers vector for segment p3->p4
 *          a            core parameter
 *          MU           shear modulus
 *          NU           poisson ratio
 *          seg12Local   1 if either node of segment p1->p2 is local to
 *                       the current domain, zero otherwise.
 *          seg34Local   1 if either node of segment p3->p4 is local to
 *                       the current domain, zero otherwise.
 *          fp1*, fp2*,  pointers to locations in which to return forces
 *          fp3*, fp4*   on nodes p1 thru p4 respectively
 *
 *
 *Refer to Appendix A.2. in "Enabling Strain Hardening Simulations with
 *Dislocation Dynamics" by A. Arsenlis et al. for mathematical treatment
 *-----------------------------------------------------------------------*/
__device__ void SpecialSegSegForce(double p1x, double p1y, double p1z,
                                double p2x, double p2y, double p2z,
                                double p3x, double p3y, double p3z,
                                double p4x, double p4y, double p4z,
                                double bpx, double bpy, double bpz,
                                double bx, double by, double bz,
                                double a, double MU, double NU, double ecrit,
                                int seg12Local, int seg34Local,
                                double *fp1x, double *fp1y, double *fp1z,
                                double *fp2x, double *fp2y, double *fp2z,
                                double *fp3x, double *fp3y, double *fp3z,
                                double *fp4x, double *fp4y, double *fp4z)
{
        double eps, c, a2, d2, flip;
        double Rx, Ry, Rz, Rdtp;
        //double Rdt;
        double oneoverL, oneoverLp;
        double temp, tempx, tempy, tempz, tempx2, tempy2, tempz2;
        double common1, common4;
        double common2x, common2y, common2z;
        double common3x, common3y, common3z;
        //double p1modx, p1mody, p1modz;
        //double p2modx, p2mody, p2modz;
        double p3modx, p3mody, p3modz;
        double p4modx, p4mody, p4modz;
        double vec1x, vec1y, vec1z;
        double tx, ty, tz;
        double tpx, tpy, tpz;
        double diffx, diffy, diffz, magdiff;
        double ndx, ndy, ndz;
        double wx, wy, wz;
        double qx, qy, qz;
        double ya, yb, za, zb;
        double fp1xcor, fp1ycor, fp1zcor;
        double fp2xcor, fp2ycor, fp2zcor;
        //double fp3xcor, fp3ycor, fp3zcor;
        //double fp4xcor, fp4ycor, fp4zcor;
        double f_003,  f_103,  f_013,  f_113,  f_213,  f_123,  f_005,  f_105;
        double f_003a, f_103a, f_013a, f_113a, f_213a, f_123a, f_005a, f_105a;
        double f_015,  f_115,  f_215,  f_125;
        double f_015a, f_115a, f_215a, f_125a;
        double Fint_003, Fint_113, Fint_005, Fint_115;
        double I_003x, I_003y, I_003z;
        double I_113x, I_113y, I_113z;
        double I_005x, I_005y, I_005z;
        double I_115x, I_115y, I_115z;
        double m4p, m8p, m4pn, a2m4pn, a2m8p;
        //double tdb, tdbp, nddb;
        //double bctx, bcty, bctz;
        //double bpctx, bpcty, bpctz;
        //double ndctx, ndcty, ndctz;
        //double bpctdb, double bpctdnd;
        //double bpctctx, bpctcty, bpctctz;
        double tpdb, tpdbp, nddbp;
        double bctpx, bctpy, bctpz;
        double bpctpx, bpctpy, bpctpz;
        double ndctpx, ndctpy, ndctpz;
        double bctpdbp, bctpdnd;
        double bctpctpx, bctpctpy, bctpctpz;
        double diffMag2;
        //double p1modMag2, p2modMag2;
        double p3modMag2, p4modMag2;
        double cotanthetac;
        double pivalue=3.141592653589793;


        cotanthetac = sqrt((1 - ecrit*1.01) / (ecrit*1.01));
        
        /* bunch of pre-factors that appear in equations to avoid
         * rewriting them all the time */
        eps    = 1e-16;
        a2     = a*a;
        m4p    = 0.25 * MU / pivalue;
        m8p    = 0.5 * m4p;
        m4pn   = m4p / ( 1 - NU ); /* factor mu/[8*pi*(1-nu)] */
        a2m4pn = a2 * m4pn;
        a2m8p  = a2 * m8p;

        /* setting pointers to zero */
        *fp1x = 0.0;
        *fp1y = 0.0;
        *fp1z = 0.0;
            
        *fp2x = 0.0;
        *fp2y = 0.0;
        *fp2z = 0.0;
            
        *fp3x = 0.0;
        *fp3y = 0.0;
        *fp3z = 0.0;
            
        *fp4x = 0.0;
        *fp4y = 0.0;
        *fp4z = 0.0;
        
        /* calculating line vector x,y,z-components for segment 4-3*/
        vec1x = p4x - p3x;
        vec1y = p4y - p3y;
        vec1z = p4z - p3z;
        
        /* line vector magnitude */
        oneoverL = 1/sqrt(vec1x*vec1x + vec1y*vec1y + vec1z*vec1z);
        
        /* unit line vector x,y,z-components for segment 4-3*/
        tx = vec1x*oneoverL;
        ty = vec1y*oneoverL;
        tz = vec1z*oneoverL;
        
        /* doing same thing, except for segment 2-1 */
        vec1x = p2x - p1x;
        vec1y = p2y - p1y;
        vec1z = p2z - p1z;
            
        oneoverLp = 1/sqrt(vec1x*vec1x + vec1y*vec1y + vec1z*vec1z);
        
        /* unit line vector x,y,z-components for segment 2-1*/
        tpx = vec1x*oneoverLp;
        tpy = vec1y*oneoverLp;
        tpz = vec1z*oneoverLp;
        
        /*dot product of unit line vectors of segments, equivalent to cosine
         *of theta angle between the two vectors*/
        c = tx*tpx + ty*tpy + tz*tpz; 
             
        /*just making sure notation is consistent and you don't get negative
         *angles*/
        flip = 0;
        if (c < 0) {
            flip = 1;
            tempx = p2x;
            tempy = p2y;
            tempz = p2z;
            p2x = p1x;
            p2y = p1y;
            p2z = p1z;
            p1x = tempx;
            p1y = tempy;
            p1z = tempz;
            tpx = -tpx;
            tpy = -tpy;
            tpz = -tpz;
            bpx = -bpx;
            bpy = -bpy;
            bpz = -bpz;
        } 
             
/*
 *      Find f1 and f2, but only if at least one of the endpoints
 *      is local to the domain.
 */
        if (seg12Local) {
            temp = (p4x-p3x)*tpx + (p4y-p3y)*tpy + (p4z-p3z)*tpz;
             
            p4modx = p3x + temp*tpx;
            p4mody = p3y + temp*tpy;
            p4modz = p3z + temp*tpz;
             
            diffx = p4x - p4modx;
            diffy = p4y - p4mody;
            diffz = p4z - p4modz;
             
            magdiff = sqrt(diffx*diffx + diffy*diffy + diffz*diffz);

            tempx2 = (0.5 * cotanthetac) * magdiff * tpx;
            tempy2 = (0.5 * cotanthetac) * magdiff * tpy;
            tempz2 = (0.5 * cotanthetac) * magdiff * tpz;

            p3modx = p3x + 0.5*diffx + tempx2;
            p3mody = p3y + 0.5*diffy + tempy2;
            p3modz = p3z + 0.5*diffz + tempz2;
             
            p4modx = p4modx + 0.5*diffx - tempx2;
            p4mody = p4mody + 0.5*diffy - tempy2;
            p4modz = p4modz + 0.5*diffz - tempz2;
             
            Rx = (p3modx - p1x);
            Ry = (p3mody - p1y);
            Rz = (p3modz - p1z);
             
            Rdtp = Rx*tpx + Ry*tpy + Rz*tpz;
             
            ndx = Rx - Rdtp*tpx;
            ndy = Ry - Rdtp*tpy;
            ndz = Rz - Rdtp*tpz;
             
            d2 = ndx*ndx + ndy*ndy + ndz*ndz;
             
            yb = p4modx*tpx + p4mody*tpy + p4modz*tpz;
            ya = p3modx*tpx + p3mody*tpy + p3modz*tpz;
            za = -(p1x*tpx + p1y*tpy + p1z*tpz);
            zb = -(p2x*tpx + p2y*tpy + p2z*tpz);
             
            SpecialSegSegForceIntegrals(a2, d2, ya, za,
                                        &f_003, &f_103, &f_013, &f_113,
                                        &f_213, &f_123, &f_005, &f_105,
                                        &f_015, &f_115, &f_215, &f_125);
        
            SpecialSegSegForceIntegrals(a2, d2, ya, zb,
                                        &f_003a, &f_103a, &f_013a, &f_113a,
                                        &f_213a, &f_123a, &f_005a, &f_105a,
                                        &f_015a, &f_115a, &f_215a, &f_125a);
        
            f_003 = f_003 - f_003a;
            f_103 = f_103 - f_103a;
            f_013 = f_013 - f_013a;
            f_113 = f_113 - f_113a;
            f_213 = f_213 - f_213a;
            f_123 = f_123 - f_123a;
            f_005 = f_005 - f_005a;
            f_105 = f_105 - f_105a;
            f_015 = f_015 - f_015a;
            f_115 = f_115 - f_115a;
            f_215 = f_215 - f_215a;
            f_125 = f_125 - f_125a;
        
            SpecialSegSegForceIntegrals(a2, d2, yb, za,
                                        &f_003a, &f_103a, &f_013a, &f_113a,
                                        &f_213a, &f_123a, &f_005a, &f_105a,
                                        &f_015a, &f_115a, &f_215a, &f_125a);
        
            f_003 = f_003 - f_003a;
            f_103 = f_103 - f_103a;
            f_013 = f_013 - f_013a;
            f_113 = f_113 - f_113a;
            f_213 = f_213 - f_213a;
            f_123 = f_123 - f_123a;
            f_005 = f_005 - f_005a;
            f_105 = f_105 - f_105a;
            f_015 = f_015 - f_015a;
            f_115 = f_115 - f_115a;
            f_215 = f_215 - f_215a;
            f_125 = f_125 - f_125a;
        
            SpecialSegSegForceIntegrals(a2, d2, yb, zb,
                                        &f_003a, &f_103a, &f_013a, &f_113a,
                                        &f_213a, &f_123a, &f_005a, &f_105a,
                                        &f_015a, &f_115a, &f_215a, &f_125a);
        
            f_003 = f_003 + f_003a;
            f_103 = f_103 + f_103a;
            f_013 = f_013 + f_013a;
            f_113 = f_113 + f_113a;
            f_213 = f_213 + f_213a;
            f_123 = f_123 + f_123a;
            f_005 = f_005 + f_005a;
            f_105 = f_105 + f_105a;
            f_015 = f_015 + f_015a;
            f_115 = f_115 + f_115a;
            f_215 = f_215 + f_215a;
            f_125 = f_125 + f_125a;
             
            tpdb = tpx*bx + tpy*by + tpz*bz;
            tpdbp = tpx*bpx + tpy*bpy + tpz*bpz;
            nddbp = ndx*bpx + ndy*bpy + ndz*bpz;
        
            bctpx = by*tpz - bz*tpy; 
            bctpy = bz*tpx - bx*tpz; 
            bctpz = bx*tpy - by*tpx;
            
            bpctpx = bpy*tpz - bpz*tpy; 
            bpctpy = bpz*tpx - bpx*tpz; 
            bpctpz = bpx*tpy - bpy*tpx;
            

            ndctpx = ndy*tpz - ndz*tpy; 
            ndctpy = ndz*tpx - ndx*tpz; 
            ndctpz = ndx*tpy - ndy*tpx;
            
            bctpdbp = bctpx*bpx + bctpy*bpy + bctpz*bpz;
            bctpdnd = bctpx*ndx + bctpy*ndy + bctpz*ndz;
            
            bctpctpx = tpdb*tpx - bx;
            bctpctpy = tpdb*tpy - by;
            bctpctpz = tpdb*tpz - bz;
            
            common1 = tpdbp*tpdb;
            
            common2x = common1*ndx;
            common2y = common1*ndy;
            common2z = common1*ndz;
            
            common3x = bctpdnd*bpctpx;
            common3y = bctpdnd*bpctpy;
            common3z = bctpdnd*bpctpz;
            
            I_003x = m4pn*(nddbp*bctpctpx+bctpdbp*ndctpx-common3x) -
                     m4p*common2x; 
            I_003y = m4pn*(nddbp*bctpctpy+bctpdbp*ndctpy-common3y) -
                     m4p*common2y; 
            I_003z = m4pn*(nddbp*bctpctpz+bctpdbp*ndctpz-common3z) -
                     m4p*common2z; 
            
            common1 = (m4pn-m4p)*tpdbp;
            
            I_113x =  common1*bctpctpx;
            I_113y =  common1*bctpctpy;
            I_113z =  common1*bctpctpz;
            
            common1 = m4pn*bctpdnd*nddbp;
            
            I_005x = -a2m8p*common2x - a2m4pn*common3x - common1*ndctpx;
            I_005y = -a2m8p*common2y - a2m4pn*common3y - common1*ndctpy;
            I_005z = -a2m8p*common2z - a2m4pn*common3z - common1*ndctpz;
            
            common1 = a2m8p*tpdbp;
            common4 = m4pn*bctpdnd*tpdbp;
            
            I_115x = -common1*bctpctpx - common4*ndctpx;
            I_115y = -common1*bctpctpy - common4*ndctpy;
            I_115z = -common1*bctpctpz - common4*ndctpz;
            
            Fint_003 = f_013 - za*f_003;
            Fint_113 = f_123 - za*f_113;
            Fint_005 = f_015 - za*f_005;
            Fint_115 = f_125 - za*f_115;
             
            *fp2x = (I_003x*Fint_003 + I_113x*Fint_113 + I_005x*Fint_005 +
                     I_115x*Fint_115) * oneoverLp;
            *fp2y = (I_003y*Fint_003 + I_113y*Fint_113 + I_005y*Fint_005 +
                     I_115y*Fint_115) * oneoverLp;
            *fp2z = (I_003z*Fint_003 + I_113z*Fint_113 + I_005z*Fint_005 +
                     I_115z*Fint_115) * oneoverLp;
             
            Fint_003 = zb*f_003 - f_013;
            Fint_113 = zb*f_113 - f_123;
            Fint_005 = zb*f_005 - f_015;
            Fint_115 = zb*f_115 - f_125;
             
            *fp1x = (I_003x*Fint_003 + I_113x*Fint_113 + I_005x*Fint_005 +
                     I_115x*Fint_115) * oneoverLp;
            *fp1y = (I_003y*Fint_003 + I_113y*Fint_113 + I_005y*Fint_005 +
                     I_115y*Fint_115) * oneoverLp;
            *fp1z = (I_003z*Fint_003 + I_113z*Fint_113 + I_005z*Fint_005 +
                     I_115z*Fint_115) * oneoverLp;
             

            diffMag2 = (diffx*diffx + diffy*diffy + diffz*diffz);
            p3modMag2 = (p3modx*p3modx + p3mody*p3mody + p3modz*p3modz);
            p4modMag2 = (p4modx*p4modx + p4mody*p4mody + p4modz*p4modz);

            if (diffMag2 > (eps * (p3modMag2+p4modMag2))) {
        
                SegSegForce(p3x, p3y, p3z, p3modx, p3mody, p3modz,
                            p1x, p1y, p1z, p2x, p2y, p2z,
                            bx, by, bz, bpx, bpy, bpz, a, MU, NU,
                            seg12Local, seg34Local,
                            &wx, &wy, &wz, &qx, &qy, &qz,
                            &fp1xcor, &fp1ycor, &fp1zcor,
                            &fp2xcor, &fp2ycor, &fp2zcor);
        
                *fp1x = *fp1x + fp1xcor;
                *fp1y = *fp1y + fp1ycor;
                *fp1z = *fp1z + fp1zcor;
                *fp2x = *fp2x + fp2xcor;
                *fp2y = *fp2y + fp2ycor;
                *fp2z = *fp2z + fp2zcor;
        
                SegSegForce(p4modx, p4mody, p4modz, p4x, p4y, p4z,
                            p1x, p1y, p1z, p2x, p2y, p2z,
                            bx, by, bz, bpx, bpy, bpz, a, MU, NU,
                            seg12Local, seg34Local,
                            &wx, &wy, &wz, &qx, &qy, &qz,
                            &fp1xcor, &fp1ycor, &fp1zcor,
                            &fp2xcor, &fp2ycor, &fp2zcor);
        
                *fp1x = *fp1x + fp1xcor;
                *fp1y = *fp1y + fp1ycor;
                *fp1z = *fp1z + fp1zcor;
                *fp2x = *fp2x + fp2xcor;
                *fp2y = *fp2y + fp2ycor;
                *fp2z = *fp2z + fp2zcor;
            }
             
/*
 *          If we flipped points 1 and 2 earlier, we have to compensate
 *          again here, but all that really needs to be switched are the
 *          forces.
 */
            if (flip == 1) {
                tempx = *fp2x;
                tempy = *fp2y;
                tempz = *fp2z;
                *fp2x = *fp1x;
                *fp2y = *fp1y;
                *fp2z = *fp1z;
                *fp1x = tempx;
                *fp1y = tempy;
                *fp1z = tempz;
            }
        } /* if segment p1->p2 is local */

        return;
}


/*-------------------------------------------------------------------------
 *
 *      Function:       SegSegForce
 *      Description:    Used to calculate the interaction forces between
 *                      dislocation segments analytically.
 *
 *      Arguments:
 *              p1*,p2*      endpoints for first dislocation segment starting
 *                           at p1x,p1y,p1z and ending at p2x,p2y,p2z
 *              p3*,p4*      endpoints for seond dislocation segment starting
 *                           at p3x,p3y,p3z and ending at p4x,p4y,p4z
 *              bxp,byp,bzp  burgers vector for segment p1 to p2
 *              bx,by,bz     burgers vector for segment p3 to p4
 *              a            core parameter
 *              MU           shear modulus
 *              NU           poisson ratio
 *              seg12Local   1 if either node of segment p1->p2 is local to
 *                           the current domain, zero otherwise.
 *              seg34Local   1 if either node of segment p3->p4 is local to
 *                           the current domain, zero otherwise.
 *              fp1*,fp2*,   pointers to locations in which to return
 *              fp3*,fp4*    forces on nodes located at p1, p2, p3 and
 *                           p4 respectively
 *            
 *Appendix A.1. in "Enabling Strain Hardening Simulations with
 *Dislocation Dynamics" by A. Arsenlis et al. for mathematical treatment          
 *-----------------------------------------------------------------------*/
__device__ void SegSegForce(double p1x, double p1y, double p1z,
                        double p2x, double p2y, double p2z,
                        double p3x, double p3y, double p3z,
                        double p4x, double p4y, double p4z,
                        double bpx, double bpy, double bpz,
                        double bx, double by, double bz,
                        double a, double MU, double NU,
                        int seg12Local, int seg34Local,
                        double *fp1x, double *fp1y, double *fp1z,
                        double *fp2x, double *fp2y, double *fp2z,
                        double *fp3x, double *fp3y, double *fp3z,
                        double *fp4x, double *fp4y, double *fp4z)
{
        double eps, d, c, c2, onemc2, onemc2inv, oneoverL, oneoverLp;
        double temp1, temp2, temp1a, temp1b, temp2a, temp2b;
        double R1x, R1y, R1z, R2x, R2y, R2z;
        double a2, m4p, m4pd, m8p, m8pd, m4pn, m4pnd, m4pnd2, m4pnd3;
        double a2m4pnd, a2m8pd, a2m4pn, a2m8p;
        double vec1x, vec1y, vec1z;
        double tx, ty, tz;
        double tpx, tpy, tpz;
        double tctpx, tctpy, tctpz;
        double ya, yb, za, zb;
        double f_003a, f_103a, f_013a, f_113a, f_203a, f_023a, f_005a, f_105a;
        double f_003,  f_103,  f_013,  f_113,  f_203,  f_023,  f_005,  f_105;
        double f_015a, f_115a, f_205a, f_025a, f_215a, f_125a, f_225a, f_305a;
        double f_015,  f_115,  f_205,  f_025,  f_215,  f_125,  f_225,  f_305;
        double f_035a, f_315a, f_135a;
        double f_035,  f_315,  f_135;
        double Fint_003, Fint_005, Fint_013, Fint_015, Fint_025, Fint_103;
        double Fint_105, Fint_115, Fint_125, Fint_205, Fint_215;
        double I_003x, I_003y, I_003z, I_005x, I_005y, I_005z;
        double I_013x, I_013y, I_013z, I_015x, I_015y, I_015z;
        double I_025x, I_025y, I_025z, I_103x, I_103y, I_103z;
        double I_105x, I_105y, I_105z, I_115x, I_115y, I_115z;
        double I_125x, I_125y, I_125z, I_205x, I_205y, I_205z;
        double I_215x, I_215y, I_215z;
        double I00ax, I00ay, I00az, I01ax, I01ay, I01az;
        double I10ax, I10ay, I10az, I00bx, I00by, I00bz;
        double I01bx, I01by, I01bz;
        //double I10bx, I10by, I10bz;
        double bctctpx, bctctpy, bctctpz;
        double bctdbp;
        double bctx, bcty, bctz;
        //double bpctpctx, bpctpcty, bpctpctz;
        double bpctpdb;
        double bpctpx, bpctpy, bpctpz;
        //double tcbpctx, tcbpcty, tcbpctz;
        //double tcbpdb;
        //double tcbpdtp;
        //double tcbpx, tcbpy, tcbpz;
        //double tctpcbpctx, tctpcbpcty, tctpcbpctz;
        double tctpcbpdb;
        //double tctpcbpdtp;
        //double tctpcbpx, tctpcbpy, tctpcbpz;
        //double tctpctx, tctpcty, tctpctz;
        double tctpdb;
        double tdb, tdbp;
        double tpcbctpx, tpcbctpy, tpcbctpz;
        double tpcbdbp;
        double tpcbdt;
        //double tpcbx, tpcby, tpcbz;
        //double tpctcbctpx, tpctcbctpy, tpctcbctpz;
        double tpctcbdbp;
        double tpctcbdt;
        //double tpctcbx, tpctcby, tpctcbz;
        double tpctctpx, tpctctpy, tpctctpz;
        double tpctdbp;
        double tpctx, tpcty, tpctz;
        double tpdb, tpdbp;
        double pivalue=3.141592653589793;

        eps = 1e-6;            

        *fp1x = 0.0;
        *fp1y = 0.0;
        *fp1z = 0.0;

        *fp2x = 0.0;
        *fp2y = 0.0;
        *fp2z = 0.0;

        *fp3x = 0.0;
        *fp3y = 0.0;
        *fp3z = 0.0;

        *fp4x = 0.0;
        *fp4y = 0.0;
        *fp4z = 0.0;

        vec1x = p4x - p3x; /*x-component of line vector of segment 4-3*/
        vec1y = p4y - p3y; /*y-component of line vector of segment 4-3*/
        vec1z = p4z - p3z; /*z-component of line vector of segment 4-3*/

        /*inverse magnitude of vector for segment 4-3*/
        oneoverL = 1/sqrt(vec1x*vec1x+vec1y*vec1y+vec1z*vec1z); 
        
        /*unit line vector x,y,z-components for segment 4-3*/
        tx = vec1x*oneoverL;
        ty = vec1y*oneoverL;
        tz = vec1z*oneoverL;
        
        vec1x = p2x - p1x; /*x-component of line vector of segment 2-1*/
        vec1y = p2y - p1y; /*y-component of line vector of segment 2-1*/
        vec1z = p2z - p1z; /*z-component of line vector of segment 2-1*/

        /*inverse magnitude of vector for segment 2-1*/
        oneoverLp = 1/sqrt(vec1x*vec1x+vec1y*vec1y+vec1z*vec1z);

        /*unit line vector x,y,z-components for segment 2-1*/
        tpx = vec1x*oneoverLp;
        tpy = vec1y*oneoverLp;
        tpz = vec1z*oneoverLp;
        
        /*tctpx, tctpy, tctpz are the determinants for submatrices of 
         *cross-product of line vectors of segments 2-1 and 4-3:
         *det|i j k ; tx ty tz ; tpx tpy tpz|*/
        tctpx = ty*tpz - tz*tpy;
        tctpy = tz*tpx - tx*tpz;
        tctpz = tx*tpy - ty*tpx;
        
        /* c is the dot product of the unit line vectors of segments 
         * 2-1 and 4-3. c2 is the magnitude of dot product of the line*/
        c = tx*tpx + ty*tpy + tz*tpz;
        c2 = c*c;
        onemc2 = 1 - c2;
        
        /*In order to check whether the two segments are parallel or not
         *an "if" statemement is used comparing the eps, an arbitrarily
         *small number close to zero, to the onemc2 variable. If the latter
         *is larger than eps, it will treat it as non-parallel and run this
         *loop, otherwise (else statement found at the bottom of script) 
         *it will use "SpecialSegSegForces" function which is defined above*/
        if (onemc2 > eps) {

            onemc2inv = 1/onemc2;

            R1x = p3x - p1x;
            R1y = p3y - p1y;
            R1z = p3z - p1z;

            R2x = p4x - p2x;
            R2y = p4y - p2y;
            R2z = p4z - p2z;

            d = (R2x*tctpx + R2y*tctpy + R2z*tctpz) * onemc2inv;

            temp1a = R1x*tx + R1y*ty + R1z*tz;
            temp1b = R2x*tx + R2y*ty + R2z*tz;

            temp2a = R1x*tpx + R1y*tpy + R1z*tpz;
            temp2b = R2x*tpx + R2y*tpy + R2z*tpz;

            ya = (temp1a - c*temp2a) * onemc2inv;
            yb = (temp1b - c*temp2b) * onemc2inv;

            za = (temp2a - c*temp1a) * onemc2inv;
            zb = (temp2b - c*temp1b) * onemc2inv;

/*
 *          For this first call to SegSegForceIntegrals() we can
 *          just pass the addresses of f_nnn variables rather than use
 *          the f_nnna variables and then copy the values.
 */
            
            /* ~f(x3-x2)? */
            SegSegForceIntegrals(a, d, c, ya, za, &f_003, &f_103,
                                 &f_013, &f_113, &f_203, &f_023,
                                 &f_005, &f_105, &f_015, &f_115,
                                 &f_205, &f_025, &f_215, &f_125,
                                 &f_225, &f_305, &f_035, &f_315,
                                 &f_135);
            /* ~f(x3-x2)? */
            SegSegForceIntegrals(a, d, c, ya, zb, &f_003a, &f_103a,
                                 &f_013a, &f_113a, &f_203a, &f_023a,
                                 &f_005a, &f_105a, &f_015a, &f_115a,
                                 &f_205a, &f_025a, &f_215a, &f_125a,
                                 &f_225a, &f_305a, &f_035a, &f_315a,
                                 &f_135a);

            f_003 = f_003 - f_003a;
            f_103 = f_103 - f_103a;
            f_013 = f_013 - f_013a;
            f_113 = f_113 - f_113a;
            f_203 = f_203 - f_203a;
            f_023 = f_023 - f_023a;
            f_005 = f_005 - f_005a;
            f_105 = f_105 - f_105a;
            f_015 = f_015 - f_015a;
            f_115 = f_115 - f_115a;
            f_205 = f_205 - f_205a;
            f_025 = f_025 - f_025a;
            f_215 = f_215 - f_215a;
            f_125 = f_125 - f_125a;
            f_225 = f_225 - f_225a;
            f_305 = f_305 - f_305a;
            f_035 = f_035 - f_035a;
            f_315 = f_315 - f_315a;
            f_135 = f_135 - f_135a;        

            /* ~f(x4-x1)? */
            SegSegForceIntegrals(a, d, c, yb, za, &f_003a, &f_103a,
                                 &f_013a, &f_113a, &f_203a, &f_023a,
                                 &f_005a, &f_105a, &f_015a, &f_115a,
                                 &f_205a, &f_025a, &f_215a, &f_125a,
                                 &f_225a, &f_305a, &f_035a, &f_315a,
                                 &f_135a); 
            

            f_003 = f_003 - f_003a;
            f_103 = f_103 - f_103a;
            f_013 = f_013 - f_013a;
            f_113 = f_113 - f_113a;
            f_203 = f_203 - f_203a;
            f_023 = f_023 - f_023a;
            f_005 = f_005 - f_005a;
            f_105 = f_105 - f_105a;
            f_015 = f_015 - f_015a;
            f_115 = f_115 - f_115a;
            f_205 = f_205 - f_205a;
            f_025 = f_025 - f_025a;
            f_215 = f_215 - f_215a;
            f_125 = f_125 - f_125a;
            f_225 = f_225 - f_225a;
            f_305 = f_305 - f_305a;
            f_035 = f_035 - f_035a;
            f_315 = f_315 - f_315a;
            f_135 = f_135 - f_135a;

            /* ~f(x3-x1)? */
            SegSegForceIntegrals(a, d, c, yb, zb, &f_003a, &f_103a,
                                 &f_013a, &f_113a, &f_203a, &f_023a,
                                 &f_005a, &f_105a, &f_015a, &f_115a,
                                 &f_205a, &f_025a, &f_215a, &f_125a,
                                 &f_225a, &f_305a, &f_035a, &f_315a,
                                 &f_135a);

            f_003 = f_003 + f_003a;
            f_103 = f_103 + f_103a;
            f_013 = f_013 + f_013a;
            f_113 = f_113 + f_113a;
            f_203 = f_203 + f_203a;
            f_023 = f_023 + f_023a;
            f_005 = f_005 + f_005a;
            f_105 = f_105 + f_105a;
            f_015 = f_015 + f_015a;
            f_115 = f_115 + f_115a;
            f_205 = f_205 + f_205a;
            f_025 = f_025 + f_025a;
            f_215 = f_215 + f_215a;
            f_125 = f_125 + f_125a;
            f_225 = f_225 + f_225a;
            f_305 = f_305 + f_305a;
            f_035 = f_035 + f_035a;
            f_315 = f_315 + f_315a;
            f_135 = f_135 + f_135a;


            a2 = a*a;
            m4p = 0.25 * MU / pivalue;
            m4pd =  m4p * d;
            m8p = 0.5 * m4p;
            m8pd = m8p * d;
            m4pn = m4p / ( 1 - NU );
            m4pnd = m4pn * d;
            m4pnd2 = m4pnd * d;
            m4pnd3 = m4pnd2 * d;
            a2m4pnd = a2 * m4pnd;
            a2m8pd = a2 * m8pd;
            a2m4pn = a2 * m4pn;
            a2m8p = a2 * m8p;

            tpctx = -tctpx;
            tpcty = -tctpy;
            tpctz = -tctpz;

            //tcbpx = ty*bpz - tz*bpy;
            //tcbpy = tz*bpx - tx*bpz;
            //tcbpz = tx*bpy - ty*bpx;

            //tpcbx = tpy*bz - tpz*by;
            //tpcby = tpz*bx - tpx*bz;
            //tpcbz = tpx*by - tpy*bx;

            bctx = by*tz - bz*ty;
            bcty = bz*tx - bx*tz;
            bctz = bx*ty - by*tx;


            bpctpx = bpy*tpz - bpz*tpy;
            bpctpy = bpz*tpx - bpx*tpz;
            bpctpz = bpx*tpy - bpy*tpx;

            tdb = tx*bx + ty*by + tz*bz;
            tdbp = tx*bpx + ty*bpy + tz*bpz;
            tpdb = tpx*bx + tpy*by + tpz*bz;
            tpdbp = tpx*bpx + tpy*bpy + tpz*bpz;

            tctpdb =  tctpx*bx + tctpy*by + tctpz*bz;
            tpctdbp = tpctx*bpx + tpcty*bpy + tpctz*bpz;
            //tcbpdtp = tpctdbp;
            tpcbdt = tctpdb;

            bpctpdb = bpctpx*bx + bpctpy*by + bpctpz*bz;
            bctdbp = bctx*bpx + bcty*bpy + bctz*bpz;
            //tcbpdb = bctdbp;
            tpcbdbp = bpctpdb;

            //tctpctx = tpx - c*tx;
            //tctpcty = tpy - c*ty;
            //tctpctz = tpz - c*tz;


            tpctctpx = tx - c*tpx;
            tpctctpy = ty - c*tpy;
            tpctctpz = tz - c*tpz;

            //tctpcbpx = tdbp*tpx - tpdbp*tx;
            //tctpcbpy = tdbp*tpy - tpdbp*ty;
            //tctpcbpz = tdbp*tpz - tpdbp*tz;

            //tpctcbx = tpdb*tx - tdb*tpx;
            //tpctcby = tpdb*ty - tdb*tpy;
            //tpctcbz = tpdb*tz - tdb*tpz;

            //tcbpctx = bpx - tdbp*tx;
            //tcbpcty = bpy - tdbp*ty;
            //tcbpctz = bpz - tdbp*tz;

            tpcbctpx = bx - tpdb*tpx;
            tpcbctpy = by - tpdb*tpy;
            tpcbctpz = bz - tpdb*tpz;
   
            //bpctpctx = tdbp*tpx - c*bpx;
            //bpctpcty = tdbp*tpy - c*bpy;
            //bpctpctz = tdbp*tpz - c*bpz;

            bctctpx = tpdb*tx - c*bx;
            bctctpy = tpdb*ty - c*by;
            bctctpz = tpdb*tz - c*bz;

            //tctpcbpctx = tdbp*tpctx;
            //tctpcbpcty = tdbp*tpcty;
            //tctpcbpctz = tdbp*tpctz;

            //tpctcbctpx = tpdb*tctpx;
            //tpctcbctpy = tpdb*tctpy;
            //tpctcbctpz = tpdb*tctpz;

            //tctpcbpdtp = tdbp - tpdbp*c;
            tpctcbdt = tpdb - tdb*c;
            tctpcbpdb =  tdbp*tpdb - tpdbp*tdb;
            tpctcbdbp = tctpcbpdb;
/*
 *          Only calculate the forces for segment p1->p2 if at least one
 *          of the segment's nodes is local to the current domain.
 */
            if (seg12Local) {

                temp1 = tpdb*tdbp + tpctcbdbp;

                I00ax = temp1 * tctpx;
                I00ay = temp1 * tctpy;
                I00az = temp1 * tctpz;

                I00bx = bpctpx * tpctcbdt;
                I00by = bpctpy * tpctcbdt;
                I00bz = bpctpz * tpctcbdt;

                temp1 = m4pnd * tpctdbp;
                temp2 = m4pnd * bctdbp;

                I_003x = m4pd*I00ax - m4pnd*I00bx + temp1*bctctpx +
                         temp2*tpctctpx;
                I_003y = m4pd*I00ay - m4pnd*I00by + temp1*bctctpy +
                         temp2*tpctctpy;
                I_003z = m4pd*I00az - m4pnd*I00bz + temp1*bctctpz +
                         temp2*tpctctpz;

                temp1 = m4pnd3 * tpctcbdt * tpctdbp;

                I_005x = a2m8pd*I00ax - a2m4pnd*I00bx - temp1*tpctctpx; 
                I_005y = a2m8pd*I00ay - a2m4pnd*I00by - temp1*tpctctpy; 
                I_005z = a2m8pd*I00az - a2m4pnd*I00bz - temp1*tpctctpz; 

                I01ax = tpctx*tpcbdbp - tpcbctpx*tdbp;
                I01ay = tpcty*tpcbdbp - tpcbctpy*tdbp;
                I01az = tpctz*tpcbdbp - tpcbctpz*tdbp;

                I01bx = -bpctpx * tpcbdt;
                I01by = -bpctpy * tpcbdt;
                I01bz = -bpctpz * tpcbdt;

                temp1 = m4pn * tpdbp;

                I_013x = -temp1 * bctctpx + m4p*I01ax - m4pn*I01bx;
                I_013y = -temp1 * bctctpy + m4p*I01ay - m4pn*I01by;
                I_013z = -temp1 * bctctpz + m4p*I01az - m4pn*I01bz;

                temp1 = m4pnd2 * (tpcbdt*tpctdbp + tpctcbdt*tpdbp);

                I_015x = a2m8p*I01ax - a2m4pn*I01bx + temp1*tpctctpx;
                I_015y = a2m8p*I01ay - a2m4pn*I01by + temp1*tpctctpy;
                I_015z = a2m8p*I01az - a2m4pn*I01bz + temp1*tpctctpz;

                I10ax = bctctpx*tdbp - tpctx*bctdbp;
                I10ay = bctctpy*tdbp - tpcty*bctdbp;
                I10az = bctctpz*tdbp - tpctz*bctdbp;

                temp1 = m4pn * tdbp; 
                temp2 = m4pn * bctdbp;

                I_103x = m4p*I10ax - temp1*bctctpx + temp2*tpctx;
                I_103y = m4p*I10ay - temp1*bctctpy + temp2*tpcty;
                I_103z = m4p*I10az - temp1*bctctpz + temp2*tpctz;

                temp1 = m4pnd2 * tpctcbdt * tdbp;
                temp2 = m4pnd2 * tpctcbdt * tpctdbp;

                I_105x = a2m8p*I10ax + temp1*tpctctpx - temp2*tpctx;
                I_105y = a2m8p*I10ay + temp1*tpctctpy - temp2*tpcty;
                I_105z = a2m8p*I10az + temp1*tpctctpz - temp2*tpctz;

                temp1 = (m4pnd * tpcbdt * tpdbp);

                I_025x = -temp1 * tpctctpx;
                I_025y = -temp1 * tpctctpy;
                I_025z = -temp1 * tpctctpz;

                temp1 = (m4pnd * tpctcbdt * tdbp);

                I_205x = temp1 * tpctx;
                I_205y = temp1 * tpcty;
                I_205z = temp1 * tpctz;

                temp1 = m4pnd * (tpctcbdt*tpdbp + tpcbdt*tpctdbp);
                temp2 = m4pnd * tpcbdt * tdbp;

                I_115x = temp1*tpctx - temp2*tpctctpx;
                I_115y = temp1*tpcty - temp2*tpctctpy;
                I_115z = temp1*tpctz - temp2*tpctctpz;

                temp1 = (m4pn * tpcbdt * tpdbp);

                I_125x = -temp1 * tpctx;
                I_125y = -temp1 * tpcty;
                I_125z = -temp1 * tpctz;

                temp1 = (m4pn * tpcbdt * tdbp);

                I_215x = -temp1 * tpctx;
                I_215y = -temp1 * tpcty;
                I_215z = -temp1 * tpctz;

                Fint_003 = f_013 - zb*f_003;
                Fint_103 = f_113 - zb*f_103;
                Fint_013 = f_023 - zb*f_013;
                Fint_005 = f_015 - zb*f_005;
                Fint_105 = f_115 - zb*f_105;
                Fint_015 = f_025 - zb*f_015;
                Fint_115 = f_125 - zb*f_115;
                Fint_205 = f_215 - zb*f_205;
                Fint_025 = f_035 - zb*f_025;
                Fint_215 = f_225 - zb*f_215;
                Fint_125 = f_135 - zb*f_125;

                *fp1x = (I_003x*Fint_003 + I_103x*Fint_103 + I_013x*Fint_013 +
                         I_005x*Fint_005 + I_105x*Fint_105 + I_015x*Fint_015 +
                         I_115x*Fint_115 + I_205x*Fint_205 + I_025x*Fint_025 +
                         I_215x*Fint_215 + I_125x*Fint_125) * oneoverLp;

                *fp1y = (I_003y*Fint_003 + I_103y*Fint_103 + I_013y*Fint_013 +
                         I_005y*Fint_005 + I_105y*Fint_105 + I_015y*Fint_015 +
                         I_115y*Fint_115 + I_205y*Fint_205 + I_025y*Fint_025 +
                         I_215y*Fint_215 + I_125y*Fint_125) * oneoverLp;

                *fp1z = (I_003z*Fint_003 + I_103z*Fint_103 + I_013z*Fint_013 +
                         I_005z*Fint_005 + I_105z*Fint_105 + I_015z*Fint_015 +
                         I_115z*Fint_115 + I_205z*Fint_205 + I_025z*Fint_025 +
                         I_215z*Fint_215 + I_125z*Fint_125) * oneoverLp;
   
                Fint_003 = za*f_003 - f_013;
                Fint_103 = za*f_103 - f_113;
                Fint_013 = za*f_013 - f_023;
                Fint_005 = za*f_005 - f_015;
                Fint_105 = za*f_105 - f_115;
                Fint_015 = za*f_015 - f_025;
                Fint_115 = za*f_115 - f_125;
                Fint_205 = za*f_205 - f_215;
                Fint_025 = za*f_025 - f_035;
                Fint_215 = za*f_215 - f_225;
                Fint_125 = za*f_125 - f_135;

                *fp2x = (I_003x*Fint_003 + I_103x*Fint_103 + I_013x*Fint_013 +
                         I_005x*Fint_005 + I_105x*Fint_105 + I_015x*Fint_015 +
                         I_115x*Fint_115 + I_205x*Fint_205 + I_025x*Fint_025 +
                         I_215x*Fint_215 + I_125x*Fint_125) * oneoverLp;

                *fp2y = (I_003y*Fint_003 + I_103y*Fint_103 + I_013y*Fint_013 +
                         I_005y*Fint_005 + I_105y*Fint_105 + I_015y*Fint_015 +
                         I_115y*Fint_115 + I_205y*Fint_205 + I_025y*Fint_025 +
                         I_215y*Fint_215 + I_125y*Fint_125) * oneoverLp;

                *fp2z = (I_003z*Fint_003 + I_103z*Fint_103 + I_013z*Fint_013 +
                         I_005z*Fint_005 + I_105z*Fint_105 + I_015z*Fint_015 +
                         I_115z*Fint_115 + I_205z*Fint_205 + I_025z*Fint_025 +
                         I_215z*Fint_215 + I_125z*Fint_125) * oneoverLp;
   
            } /* if segment p1->p2 is "local" */

        } else {
/*
 *          The two lines are parallel, so we have to use a special
 *          lower dimensional function
 */
            SpecialSegSegForce(p1x, p1y, p1z, p2x, p2y, p2z,
                               p3x, p3y, p3z, p4x, p4y, p4z,
                               bpx, bpy, bpz, bx, by, bz, a, MU, NU,
                               eps, seg12Local, seg34Local,
                               fp1x, fp1y, fp1z, fp2x, fp2y, fp2z,
                               fp3x, fp3y, fp3z, fp4x, fp4y, fp4z);
       }

       return;
}


/* --------------------------------------------------------------*/
/* ------------ the MEX driver runs on the CPU ------------------*/
/* --------------------------------------------------------------*/

/*
void mexFunction (int nlhs, mxArray *plhs[],
                  int nrhs, const mxArray *prhs[]) 
{
    //Get the total memory needed for the various arrays on the GPU (bytes)
    mwSize m = mxGetM(prhs[0]);
    mwSize n = mxGetN(prhs[0]);
    mwSize arraySize = m * n;
    mem_size = sizeof(double) * arraySize;
    //all the other arrays are the same size (except a,MU,NU,linkid,S)
    
    // Define variables
    double *P1x_vec, *P1y_vec, *P1z_vec;
    double *P2x_vec, *P2y_vec, *P2z_vec;
    double *Bx_vec, *By_vec, *Bz_vec;
    double a,MU,NU;
    int linkid,S;
    double *f0x, *f0y, *f0z, *f1x, *f1y, *f1z;

    // allocate the memory on the GPU
    // memory for input vectors
    HANDLE_ERROR ( cudaMalloc( &P1x_vec, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &P1y_vec, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &P1z_vec, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &P2x_vec, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &P2y_vec, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &P2z_vec, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &Bx_vec, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &By_vec, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &Bz_vec, mem_size ) );
    // memory for constants
    HANDLE_ERROR ( cudaMalloc( &a, sizeof(double) ) );
    HANDLE_ERROR ( cudaMalloc( &MU, sizeof(double) ) );
    HANDLE_ERROR ( cudaMalloc( &NU, sizeof(double) ) );
    HANDLE_ERROR ( cudaMalloc( &linkid, sizeof(int) ) );
    HANDLE_ERROR ( cudaMalloc( &S, sizeof(int) ) );
    // memory for output vectors
    HANDLE_ERROR ( cudaMalloc( &f0x, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &f0y, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &f0z, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &f1x, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &f1y, mem_size ) );
    HANDLE_ERROR ( cudaMalloc( &f1z, mem_size ) );
    
    // Copy input data across to the GPU
    // memory for input vectors
    HANDLE_ERROR ( cudaMemcpy( P1x_vec, (double*) mexGetData(prhs[0]), mem_size, cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( P1y_vec, (double*) mexGetData(prhs[1]), mem_size, cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( P1z_vec, (double*) mexGetData(prhs[2]), mem_size, cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( P2x_vec, (double*) mexGetData(prhs[3]), mem_size, cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( P2y_vec, (double*) mexGetData(prhs[4]), mem_size, cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( P2z_vec, (double*) mexGetData(prhs[5]), mem_size, cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( Bx_vec, (double*) mexGetData(prhs[6]), mem_size, cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( By_vec, (double*) mexGetData(prhs[7]), mem_size, cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( Bz_vec, (double*) mexGetData(prhs[8]), mem_size, cudaMemcpyHostToDevice ) );
    // memory for constants
    HANDLE_ERROR ( cudaMemcpy( a, (double) mexGetData(prhs[9]), sizeof(double), cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( MU, (double) mexGetData(prhs[10]), sizeof(double), cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( NU, (double) mexGetData(prhs[11]), sizeof(double), cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( linkid, (int) mexGetData(prhs[12]), sizeof(int), cudaMemcpyHostToDevice ) );
    HANDLE_ERROR ( cudaMemcpy( S, (int) mexGetData(prhs[13]), sizeof(int), cudaMemcpyHostToDevice ) );
    
    // Define the block and grid size - make this dynamic at later stage !!!!
    int blockSize = 1024;
    dim3 block(blockSize);
    dim 3 grid(ceil(arraySize/(double)blockSize));
    
    // Use the CUDA runtime to run the kernel
    SegForceNBodyCUDA <<< grid, block >>> (P1x_vec, P1y_vec, P1z_vec,
                                           P2x_vec, P2y_vec, P2z_vec,
                                           Bx_vec, By_vec, Bz_vec,
                                           a, MU, NU,
                                           linkid, S,
                                           f0x, f0y, f0z,
                                           f1x, f1y, f1z);
                                           
    // Create the output arrays for MATLAB
    plhs[0]=mxCreateNumericMatrix(m,n,mxSINGLE_CLASS,mxREAL);
    plhs[1]=mxCreateNumericMatrix(m,n,mxSINGLE_CLASS,mxREAL);
    plhs[2]=mxCreateNumericMatrix(m,n,mxSINGLE_CLASS,mxREAL);
    plhs[3]=mxCreateNumericMatrix(m,n,mxSINGLE_CLASS,mxREAL);
    plhs[4]=mxCreateNumericMatrix(m,n,mxSINGLE_CLASS,mxREAL);
    plhs[5]=mxCreateNumericMatrix(m,n,mxSINGLE_CLASS,mxREAL);

    //Copy the data from the card into the MATLAB array
    cudaMemcpy( (double*)mxGetData(plhs[0]), f0x, mem_size, cudaMemcpyDeviceToHost );
    cudaMemcpy( (double*)mxGetData(plhs[1]), f0y, mem_size, cudaMemcpyDeviceToHost );
    cudaMemcpy( (double*)mxGetData(plhs[2]), f0z, mem_size, cudaMemcpyDeviceToHost );
    cudaMemcpy( (double*)mxGetData(plhs[3]), f1x, mem_size, cudaMemcpyDeviceToHost );
    cudaMemcpy( (double*)mxGetData(plhs[4]), f1y, mem_size, cudaMemcpyDeviceToHost );
    cudaMemcpy( (double*)mxGetData(plhs[5]), f1z, mem_size, cudaMemcpyDeviceToHost );

    //Free the data on the card
    cudaFree( P1x_vec );
    cudaFree( P1y_vec );
    cudaFree( P1z_vec );
    cudaFree( P2x_vec );
    cudaFree( P2y_vec );
    cudaFree( P2z_vec );
    cudaFree( Bx_vec );
    cudaFree( By_vec );
    cudaFree( Bz_vec );
    cudaFree( a );
    cudaFree( MU );
    cudaFree( NU );
    cudaFree( linkid );
    cudaFree( S );
}
*/

    
    
